#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <glad/glad.h>
#include <glfw3.h>
#include <string.h>
#include <fstream>
#include <sstream>

#define fov 1.0f
#define scr_w 512
#define scr_h 512

#define num_triangles 1
#define max_streams 512

hipError_t ercall;
#define CCALL(call) ercall = call; if(hipSuccess != ercall){fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", __FILE__, __LINE__, hipGetErrorString(ercall)); exit(EXIT_FAILURE);}
#define printLastErrorCUDA() printf(hipGetErrorString(hipGetLastError()))

#define dot(vec3_v1, vec3_v2) (vec3_v1.x * vec3_v2.x + vec3_v1.y * vec3_v2.y + vec3_v1.z * vec3_v2.z)
#define dot2D(vec2_v1, vec2_v2) (vec2_v1.x * vec2_v2.x + vec2_v1.y * vec2_v2.y)
#define matrix2D_eval(float_a , float_b, float_c, float_d) (float_a*float_d - float_b*float_c)
#define matgnitude(vec3_a) (sqrtf(dot(vec3_a, vec3_a)))
#define magnitude2D(vec2_a) (sqrtf(dot2D(vec2_a, vec2_a)))


// init of structs and methods as well as global vars and respective functions and macros
//*****************************************************************************************************************************************************************************************

// too lazy to set up cudas rng so i use this bad one
inline __host__ __device__ long int xorRand(unsigned int seed) {
    seed ^= seed << 13;
    seed ^= seed >> 17;
    seed ^= seed << 5;
    return seed;
}

struct vec2 {
    float x, y;

    __host__ __device__ vec2() : x(0.0f), y(0.0f) {}
    __host__ __device__ vec2(float X, float Y) : x(X), y(Y) {}

    inline __host__ __device__ vec2 operator+(const vec2& f) const {
        return vec2(x + f.x, y + f.y);
    }

    inline __host__ __device__ vec2 operator-(const vec2& f) const {
        return vec2(x - f.x, y - f.y);
    }

    inline __host__ __device__ vec2 operator*(const float scalar) const {
        return vec2(x * scalar, y * scalar);
    }

    inline __host__ __device__ vec2 normalize() {
        const float scl = magnitude2D((*this));
        return vec2(x / scl, y / scl);
    }
};

// Define the vec3 struct
struct vec3 {
    float x, y, z;

    __host__ __device__ vec3() : x(0), y(0), z(0) {}
    __host__ __device__ vec3(float x, float y, float z) : x(x), y(y), z(z) {}

    inline __host__ __device__ vec3 operator+(const vec3& f) const {
        return vec3(x + f.x, y + f.y, z + f.z);
    }

    inline __host__ __device__ vec3 operator-(const vec3& f) const {
        return vec3(x - f.x, y - f.y, z - f.z);
    }

    inline __host__ __device__ vec3 operator*(const float scalar) const {
        return vec3(x * scalar, y * scalar, z * scalar);
    }

    inline __host__ __device__ vec3 normalize() {
        const float scl = matgnitude((*this));
        return vec3(x / scl, y / scl, z / scl);
    }

    inline __host__ __device__ vec2 convert_vec2() {
        return vec2(x / (z * fov), y / (z * fov));
    }
};

// cross is more logical as its own function

inline __host__ __device__ vec3 cross(const vec3 v1, const vec3 v2) {
    vec3 ret;
    ret.x = matrix2D_eval(v1.y, v1.z, v2.y, v2.z);
    ret.y = matrix2D_eval(v1.x, v1.z, v2.x, v2.z);
    ret.z = matrix2D_eval(v1.x, v1.y, v2.x, v2.y);
    return ret;
}

inline __host__ __device__ float get_max(float f1, float f2) {
    return (f1 > f2) * f1 + (f1 <= f2) * f2;
}

inline __host__ __device__ float get_min(float f1, float f2) {
    return (f1 < f2) * f1 + (f1 >= f2) * f2;
}

struct bounding_box{
    vec2 min, max;

    __host__ __device__ bounding_box(){}

    __host__ __device__ bounding_box(const float minx, const float maxx, const float miny, const float maxy) : min(vec2(minx, miny)), max(vec2(maxx, maxy)){}
};

struct triangle2D {
    vec2 p1, p2, p3;
    float denom, y2_y3, x1_x3, x3_x2, y3_y1;
    bounding_box bound_box;

    __host__ __device__ triangle2D() {}

    inline __host__ __device__ void calc_denom_and_vals() {
        denom = (p2.y - p3.y) * (p1.x - p3.x) + (p3.x - p2.x) * (p1.y - p3.y);
        y2_y3 = p2.y - p3.y;
        x1_x3 = p1.x - p3.x;
        x3_x2 = p3.x - p2.x;
        y3_y1 = p3.y - p1.y;
        float minx = get_min(p1.x, p2.x);
        minx = get_min(minx, p3.x);

        float maxx = get_max(p1.x, p2.x);
        maxx = get_max(maxx, p3.x);

        float miny = get_min(p1.y, p2.y);
        miny = get_min(miny, p3.y);

        float maxy = get_max(p1.y, p2.y);
        maxy = get_max(maxy, p3.y);

        bound_box = bounding_box(minx, maxx, miny, maxy);
    }

    __host__ __device__ triangle2D(const vec2 P1, const vec2 P2, const vec2 P3) {
        p1 = P1; p2 = P2; p3 = P3;
        calc_denom_and_vals();
    }

    inline __host__ __device__ bool point_in_triangle(const vec2 p) {
        const float x3m = p.x - p3.x;
        const float y3m = p.y - p3.y;

        const float a = (y2_y3 * x3m + x3_x2 * y3m) / denom;
        const float b = (y3_y1 * x3m + x1_x3 * y3m) / denom;
        const float c = 1.0f - a - b;
        return 0 <= a && a <= 1 && 0 <= b && b <= 1 && 0 <= c && c <= 1;
    }
};


struct triangle {
    vec3 p1, p2, p3;
    vec3 nv;
    vec3 sb21, sb31;
    float dot2121, dot2131, dot3131;

    __host__ __device__ triangle() : p1(vec3(0.0f, 0.0f, 0.0f)), p2(vec3(0.0f, 0.0f, 0.0f)), p3(vec3(0.0f, 0.0f, 0.0f)) {}

    __host__ __device__ triangle(const vec3 P1, const vec3 P2, const vec3 P3) {
        p1 = P1;
        p2 = P2;
        p3 = P3;
        sb21 = p2 - p1;
        sb31 = p3 - p1;
        dot2121 = dot(sb21, sb21);
        dot2131 = dot(sb21, sb31);
        dot3131 = dot(sb31, sb31);
        nv = cross(sb21, sb31).normalize();
    }

    inline __host__ __device__ triangle2D convert_to_2D() {
        return triangle2D(p1.convert_vec2(), p2.convert_vec2(), p3.convert_vec2());
    }
};

struct color {
    float r, g, b;

    __host__ __device__ color(){}
    __host__ __device__ color(const float R, const float G, const float B) : r(R), g(G), b(B){}

    inline __host__ __device__ color operator+(const color& c) {
        return color(r + c.r, g + c.g, b + c.b);
    }

    inline __host__ __device__ color operator*(const float f) {
        return color(r * f, g * f, b * f);
    }

    inline __host__ __device__ color operator*(const bool f) {
        return color(r * f, g * f, b * f);
    }
};

// global device screen buffer
__device__ char screen_buffer[sizeof(color) * scr_w * scr_h];

// global device array of all triangles
__device__ char triangles[sizeof(triangle) * num_triangles];

// rasterization function tests
inline __global__ void rasterize_triangles_single_thread(color c) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= num_triangles) { return; }
    triangle2D t2D = ((triangle*)triangles)[index].convert_to_2D();

    const int p_minx = (int)t2D.bound_box.min.x;
    const int p_miny = (int)t2D.bound_box.min.y;
    const int p_maxx = (int)t2D.bound_box.max.x;
    const int p_maxy = (int)t2D.bound_box.max.y;
    for(int x = p_minx; x <= p_maxx; x++) {
        for (int y = p_miny; y < p_maxy; y++) {
            ((color*)screen_buffer)[x + y * scr_w] = c * t2D.point_in_triangle(vec2(x, y));
        }
    }
}

__global__ void rasterize_triangle_multi_thread(int index, color c) {
    __shared__ triangle2D t2D;
    __shared__ vec2 min, max;

    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if (id == 0) {
        t2D = ((triangle*)triangles)[index].convert_to_2D();
        min = t2D.bound_box.min;
        max = t2D.bound_box.max;
    }
    
    const int x_dif = max.x - min.x;
    const int y_dif = max.y - min.y;

    if (id > x_dif * y_dif) { return; }

    const int x = id % x_dif + min.x;
    const int y = id / x_dif + min.y;

    ((color*)screen_buffer)[x + y * scr_w] = c * t2D.point_in_triangle(vec2(x, y));
}

int clamp(int i) {
    return (i < max_streams) ? i : max_streams;
}

// being worked on
/*
void rasterize_all_triangles_multi_thread() {
    hipStream_t streams[max_streams];

    const int num_iterations = num_triangles / max_streams + 1;

    int total_tris = 0;

    for (int i = 0; i < num_iterations; i++) {
        const int num_streams = clamp((num_triangles - total_tris));
        total_tris += num_streams;
        for (int s = 0; s < num_streams; s++) {
            hipStreamCreate(&streams[s]);
            rasterize_triangle_multi_thread<<<512, >>>(s, color(1.0f, 0.0f, 0.0f));
        }
    }
}
*/

void rasterize_all_triangles(color c) {
    rasterize_triangles_single_thread << <256, num_triangles / 256 + 1 >> > (c);
}

void add_triangle(vec3 p1, vec3 p2, vec3 p3, int idx) {
    triangle t = triangle(p1, p2, p3);
    hipMemcpyToSymbol(HIP_SYMBOL(triangles), &t, sizeof(triangle), sizeof(triangle) * idx);
}

//*****************************************************************************************************************************************************************************************
// opengl stuff
// draws 2 triangles at z=0 and textures them with the pixel colors outputted by the cuda program
// no interop, data transfers from GPU to CPU each frame

char cpu_colors[sizeof(color) * scr_w * scr_h];

void copyBufferToCPU() {
    hipMemcpyFromSymbol(cpu_colors, HIP_SYMBOL(screen_buffer), sizeof(color) * scr_w * scr_h);
}

void framebuffer_size_callback(GLFWwindow* window, int width, int height);
void processInput(GLFWwindow* window);

// settings
unsigned int SCR_WIDTH = scr_w;
unsigned int SCR_HEIGHT = scr_h;


char* vertexShaderSource = "#version 330 core\n"
"layout (location = 0) in vec3 aPos;\n"
"layout(location = 1) in vec3 aColor;\n"
"layout(location = 2) in vec2 aTexCoord;\n"
"out vec3 ourColor;\n"
"out vec2 TexCoord;\n"
"void main()\n"
"{\n"
"   gl_Position = vec4(aPos.x, aPos.y, aPos.z, 1.0);\n"
"   ourColor = aColor;\n"
"   TexCoord = vec2(aTexCoord.x, aTexCoord.y);\n"
"}\0";

char* fragmentShaderSource = "#version 330 core\n"
"out vec4 FragColor;\n"
"in vec3 ourColor;\n"
"in vec2 TexCoord;\n"
"uniform sampler2D texture1;\n"
"void main()\n"
"{\n"
"   FragColor = texture(texture1, TexCoord);\n"
"}\n\0";

float truncate(float f) {
    return fabs(1.0 / (1.0 + exp(-1.0 * f))-0.5) * 2.0f;
}

int main()
{
    // add triangles
    for (int t = 0; t < num_triangles; t++) {
        add_triangle(vec3(0.0f, 0.0f, 1.0f), vec3(100.0f, 0.0f, 1.0f), vec3(100.0f, 100.0f, 1.0f), t);
    }
    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
#ifdef __APPLE__
    glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
#endif
    GLFWmonitor* monitor = glfwGetPrimaryMonitor();
    const GLFWvidmode* mode = glfwGetVideoMode(monitor);

    GLFWwindow* window = glfwCreateWindow(scr_w, scr_h, "Cuda-openGL Interop", NULL, NULL);

    glfwMakeContextCurrent(window);
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
    gladLoadGLLoader((GLADloadproc)glfwGetProcAddress);
    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        return -1;
    }
    unsigned int vertexShader = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
    glCompileShader(vertexShader);
    // check for shader compile errors
    int success;
    char infoLog[512];
    glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
    unsigned int fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
    glCompileShader(fragmentShader);
    // check for shader compile errors
    glGetShaderiv(fragmentShader, GL_COMPILE_STATUS, &success);
    if (!success) {
        glGetProgramInfoLog(fragmentShader, 512, NULL, infoLog);
        printf("ERROR::FRAGMENT::PROGRAM::LINKING_FAILED %s\n", infoLog);
    }
    GLuint shaderProgram = glCreateProgram();
    glAttachShader(shaderProgram, vertexShader);
    glAttachShader(shaderProgram, fragmentShader);
    glLinkProgram(shaderProgram);
    // check for linking errors
    glGetProgramiv(shaderProgram, GL_LINK_STATUS, &success);
    if (!success) {
        glGetProgramInfoLog(shaderProgram, 512, NULL, infoLog);
        printf("ERROR::SHADER::PROGRAM::LINKING_FAILED %s\n", infoLog);
    }

    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    float vertices[] = {
        // positions          // colors           // texture coords
         1.0f,  1.0f, 0.0f,   1.0f, 1.0f, 1.0f,   1.0f, 1.0f, // top right
         1.0f, -1.0f, 0.0f,   1.0f, 1.0f, 1.0f,   1.0f, 0.0f, // bottom right
        -1.0f, -1.0f, 0.0f,   1.0f, 1.0f, 1.0f,   0.0f, 0.0f, // bottom left
        -1.0f,  1.0f, 0.0f,   1.0f, 1.0f, 1.0f,   0.0f, 1.0f  // top left 
    };
    unsigned int indices[] = {
        0, 1, 3, // first triangle
        1, 2, 3  // second triangle
    };
    unsigned int VBO, VAO, EBO;
    glGenVertexArrays(1, &VAO);
    glGenBuffers(1, &VBO);
    glGenBuffers(1, &EBO);

    glBindVertexArray(VAO);

    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);

	// position attribute
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);
	// color attribute
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void*)(3 * sizeof(float)));
	glEnableVertexAttribArray(1);
	// texture coord attribute
	glVertexAttribPointer(2, 2, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void*)(6 * sizeof(float)));
	glEnableVertexAttribArray(2);

    unsigned int texture1;
    //uint8_t pixels[grid_h * grid_l * 3];
    glPixelStorei(GL_UNPACK_ALIGNMENT, 1);
    glGenTextures(1, &texture1);
    glBindTexture(GL_TEXTURE_2D, texture1);
    // set the texture wrapping parameters
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);	
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST_MIPMAP_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST_MIPMAP_NEAREST);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, scr_w, scr_h, 0, GL_RGB, GL_FLOAT, cpu_colors);
	glGenerateMipmap(GL_TEXTURE_2D);
    glUseProgram(shaderProgram); 
    glUniform1i(glGetUniformLocation(shaderProgram, "texture1"), 0);

    clock_t start, end;
    int frametime = 0;
    unsigned int frame = 0;
    while (!glfwWindowShouldClose(window))
    {
        start = clock();
        processInput(window);
        glClearColor(0.0f, 0.0f, 1.0f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);
        int ind = 0;
        // **
        // dodaj boje tu u pixels
        rasterize_all_triangles(color(1.0f, 0.0f, 0.0f));
        copyBufferToCPU();
        // **
		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, scr_w, scr_h, 0, GL_RGB, GL_FLOAT, cpu_colors);
		glGenerateMipmap(GL_TEXTURE_2D);
        // bind textures on corresponding texture units
        glActiveTexture(GL_TEXTURE0);
        glBindTexture(GL_TEXTURE_2D, texture1);

        // render container
        glUseProgram(shaderProgram);
        glBindVertexArray(VAO);
        glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);

        glfwSwapBuffers(window);
        glfwPollEvents();
        end = clock();
        if (frame % 10 == 1) {
            int fps = 10000 / (frametime);
            if (fps < 10) {
                printf("\rFPS: 000%d", fps);
            }

            if (fps < 100) {
                printf("\rFPS: 00%d", fps);
            }

            else if (fps < 1000) {
                printf("\rFPS: 0%d", fps);
            }
            
            else {
                printf("\rFPS: %d", fps);
            }
            frametime = 0;
        }
        frame++;
        frametime += end - start;
    }
    glDeleteVertexArrays(1, &VAO);
    glDeleteBuffers(1, &VBO);
    glDeleteBuffers(1, &EBO);
    glfwTerminate();
    return 0;
}

void processInput(GLFWwindow* window)
{
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);
}

void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}