#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include "cuda_device_runtime_api.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <glad/glad.h>
#include <glfw3.h>
#include <string.h>
#include <fstream>
#include <sstream>
#include <iostream>
#include <unordered_map>
#include <vector>
#include <list>
#include <algorithm>
#include <hip/hip_fp16.h>
#include <thread>

#define fov 0.0035f
#define scr_w 512
#define scr_h 512

#define num_triangles 207
#define max_streams 512

hipError_t ercall;
hipError_t err;
#define CCALL(call) ercall = call; if(hipSuccess != ercall){fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", __FILE__, __LINE__, hipGetErrorString(ercall)); exit(EXIT_FAILURE);}
#define printLastErrorCUDA() err = hipGetLastError(); if(err != hipSuccess){printf("%s\n",hipGetErrorString(err));}

#define dot(vec3_v1, vec3_v2) (vec3_v1.x * vec3_v2.x + vec3_v1.y * vec3_v2.y + vec3_v1.z * vec3_v2.z)
#define dot_fastmath(vec3_v1, vec3_v2) __fmaf_rn(vec3_v1.x, vec3_v2.x,__fmaf_rn(vec3_v1.y, vec3_v2.y,vec3_v1.z * vec3_v2.z))
#define dot2D(vec2_v1, vec2_v2) (vec2_v1.x * vec2_v2.x + vec2_v1.y * vec2_v2.y)
#define matrix2D_eval(float_a , float_b, float_c, float_d) (float_a*float_d - float_b*float_c)
#define matgnitude(vec3_a) (sqrtf(dot(vec3_a, vec3_a)))
#define magnitude2D(vec2_a) (sqrtf(dot2D(vec2_a, vec2_a)))


// init of structs and methods as well as global vars and respective functions and macros
//*****************************************************************************************************************************************************************************************

// too lazy to set up cudas rng so i use this bad one
inline __host__ __device__ long int xorRand(unsigned int seed) {
    seed ^= seed << 13;
    seed ^= seed >> 17;
    seed ^= seed << 5;
    return seed;
}

struct vec2 {
    float x, y;

    __host__ __device__ vec2() : x(0.0f), y(0.0f) {}
    __host__ __device__ vec2(float X, float Y) : x(X), y(Y) {}

    inline __host__ __device__ vec2 operator+(const vec2& f) const {
        return vec2(x + f.x, y + f.y);
    }

    inline __host__ __device__ vec2 operator-(const vec2& f) const {
        return vec2(x - f.x, y - f.y);
    }

    inline __host__ __device__ vec2 operator*(const float scalar) const {
        return vec2(x * scalar, y * scalar);
    }

    inline __host__ __device__ vec2 normalize() {
        const float scl = magnitude2D((*this));
        return vec2(x / scl, y / scl);
    }

    inline __device__ float dist_from_vec(vec2& v) {
        const float addx = (x + v.x);
        const float addy = (y + v.y);
        return __fsqrt_rn(__fmaf_rn(addx,addx,addy*addy));
    }
};

// Define the vec3 struct
struct vec3 {
    float x, y, z;

    __host__ __device__ vec3() : x(0), y(0), z(0) {}
    __host__ __device__ vec3(float x, float y, float z) : x(x), y(y), z(z) {}

    inline __host__ __device__ vec3 operator+(const vec3& f) const {
        return vec3(x + f.x, y + f.y, z + f.z);
    }

    inline __host__ __device__ vec3 operator-(const vec3& f) const {
        return vec3(x - f.x, y - f.y, z - f.z);
    }

    inline __host__ __device__ vec3 operator*(const float scalar) const {
        return vec3(x * scalar, y * scalar, z * scalar);
    }

    inline __host__ __device__ vec3 normalize() {
        const float scl = matgnitude((*this));
        return vec3(x / scl, y / scl, z / scl);
    }

    inline __host__ __device__ bool operator==(const vec3& f) const {
        return fabs(x - f.x) < 0.01f && fabs(y - f.y) < 0.01f && fabs(z - f.z) < 0.01f;
    }

    inline __host__ __device__ vec2 convert_vec2() const {
        return vec2(x / (z * fov), y / (z * fov));
    }
};

// cross is more logical as its own function

inline __host__ __device__ vec3 cross(const vec3 v1, const vec3 v2) {
    vec3 ret;
    ret.x = matrix2D_eval(v1.y, v1.z, v2.y, v2.z);
    ret.y = matrix2D_eval(v1.x, v1.z, v2.x, v2.z);
    ret.z = matrix2D_eval(v1.x, v1.y, v2.x, v2.y);
    return ret;
}

struct bounding_box{
    vec2 min, max;

    __device__ bounding_box(){}

    __host__ __device__ bounding_box(const float minx, const float maxx, const float miny, const float maxy) : min(vec2(minx, miny)), max(vec2(maxx, maxy)){}
};

typedef struct {
    float a, b, c;
}barycentric_return;

struct triangle2D {
    vec2 p1, p2, p3;
    float denom, y2_y3, x1_x3, x3_x2, y3_y1;
    bounding_box bound_box;

    __host__ __device__ triangle2D() {}

    inline __host__ __device__ void calc_denom_and_vals() {
        denom = 1.0f/((p2.y - p3.y) * (p1.x - p3.x) + (p3.x - p2.x) * (p1.y - p3.y));
        y2_y3 = (p2.y - p3.y) * denom;
        x1_x3 = (p1.x - p3.x) * denom;
        x3_x2 = (p3.x - p2.x) * denom;
        y3_y1 = (p3.y - p1.y) * denom;
        float minx = fminf(p1.x, p2.x);
        minx = fminf(minx, p3.x);

        float maxx = fmaxf(p1.x, p2.x);
        maxx = fmaxf(maxx, p3.x);

        float miny = fminf(p1.y, p2.y);
        miny = fminf(miny, p3.y);

        float maxy = fmaxf(p1.y, p2.y);
        maxy = fmaxf(maxy, p3.y);

        bound_box = bounding_box(minx-1, maxx+1, miny-1, maxy+1);
    }

    __host__ __device__ triangle2D(const vec2 P1, const vec2 P2, const vec2 P3) {
        p1 = P1; p2 = P2; p3 = P3;
    }

    inline __device__ barycentric_return point_in_triangle(const vec2 p, int seed) const {
        const float x3m = p.x - p3.x;
        const float y3m = p.y - p3.y;

        barycentric_return r;
        r.a = (y2_y3 * x3m + x3_x2 * y3m); r.b = (y3_y1 * x3m + x1_x3 * y3m); r.c = 1.0f - r.a - r.b;
        return r;
    }
};


struct triangle {
    vec3 p1, p2, p3;
    vec3 nv;
    vec3 sb21, sb31;
    float dot2121, dot2131, dot3131;

    __host__ __device__ triangle() : p1(vec3(0.0f, 0.0f, 0.0f)), p2(vec3(0.0f, 0.0f, 0.0f)), p3(vec3(0.0f, 0.0f, 0.0f)) {}

    __host__ __device__ triangle(const vec3 P1, const vec3 P2, const vec3 P3) {
        p1 = P1;
        p2 = P2;
        p3 = P3;
        sb21 = p2 - p1;
        sb31 = p3 - p1;
        dot2121 = dot(sb21, sb21);
        dot2131 = dot(sb21, sb31);
        dot3131 = dot(sb31, sb31);
        nv = cross(sb21, sb31).normalize();
    }

    inline __host__ __device__ triangle2D convert_to_2D() const {
        return triangle2D(p1.convert_vec2(), p2.convert_vec2(), p3.convert_vec2());
    }
};

struct color {
    float r, g, b;

    __host__ __device__ color(){}
    __host__ __device__ color(const float R, const float G, const float B) : r(R), g(G), b(B){}

    inline __host__ __device__ color operator+(const color& c) {
        return color(r + c.r, g + c.g, b + c.b);
    }

    inline __host__ __device__ color operator*(const float f) {
        return color(r * f, g * f, b * f);
    }

    inline __host__ __device__ color operator*(const bool f) {
        return color(r * f, g * f, b * f);
    }
};

// global device screen buffer
__device__ char screen_buffer[sizeof(color) * scr_w * scr_h];

__device__ float depth_buffer[scr_w * scr_h];

// global device array of all triangles
__device__ char triangles[sizeof(triangle) * num_triangles];

// vertex norms
__constant__ char vertex_norms[sizeof(vec3) * num_triangles * 3];

typedef struct {
    vec3 v1, v2, v3;
}triplevec3;

inline __device__ float barycentric_float_interpolation(const vec3 floats, const barycentric_return r) {
    return floats.x * r.a + floats.y * r.b + floats.z * r.c;
}

__global__ void fillPixels(triangle2D t2D, const vec3 z_coords, const triplevec3 n) {
    const int id = threadIdx.x + blockIdx.x * blockDim.x;

    const int tmp = (t2D.bound_box.max.x - t2D.bound_box.min.x);

    const int tmp2 = (t2D.bound_box.max.y - t2D.bound_box.min.y);

    const int x = id % tmp + t2D.bound_box.min.x;
    const int y = id / tmp + t2D.bound_box.min.y;

    const barycentric_return r = t2D.point_in_triangle(vec2(x, y), 10 * id);

    const float scl = -1.0f / ((tmp < tmp2) * tmp + (tmp > tmp2) * tmp2);

    const float z = __fmaf_rn(z_coords.x, r.a, __fmaf_rn(z_coords.y, r.b, z_coords.z * r.c));

    float d;
    atomicExch(&d, depth_buffer[x + y * scr_w]);

    if (!(r.a >= scl && r.b >= scl && r.c >= scl && (d == 0 || d > z))) {
        return;
    }

    const vec3 interpolated_norm = vec3(__fmaf_rn(n.v1.x, r.a, __fmaf_rn(n.v2.x, r.b, n.v3.x * r.c)), __fmaf_rn(n.v1.y, r.a, __fmaf_rn(n.v2.y, r.b, n.v3.y * r.c)), __fmaf_rn(n.v1.z, r.a, __fmaf_rn(n.v2.z, r.b, n.v3.z * r.c)));
    //((color*)screen_buffer)[x + y * scr_w] = color(r.a, r.b, r.c);
    atomicExch(&(depth_buffer[x + y * scr_w]), z);
    //depth_buffer[x + y * scr_w] = z;
    //((color*)screen_buffer)[x + y * scr_w] = color(1.0f, (z-22) / 20.0f, (z-22) / 20.0f);
    ((color*)screen_buffer)[x + y * scr_w] = color(interpolated_norm.x, interpolated_norm.y, interpolated_norm.z);
}

// rasterization function tests
__global__ void rasterize_triangles_single_thread() {
    const int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= num_triangles) { return; }
    const triangle t = ((triangle*)triangles)[index];
    triangle2D t2D = t.convert_to_2D();

    const vec2 tmp = vec2(scr_w / 2, scr_h / 2);

    t2D.p1 = t2D.p1 + tmp;
    t2D.p2 = t2D.p2 + tmp;
    t2D.p3 = t2D.p3 + tmp;
    
    t2D.calc_denom_and_vals();

    __syncthreads();
    fillPixels << <32, ((int)t2D.bound_box.max.x - (int)t2D.bound_box.min.x) * ((int)t2D.bound_box.max.y - (int)t2D.bound_box.min.y) / 32 + 1 >> > (t2D, vec3(t.p1.z, t.p2.z, t.p3.z), ((triplevec3*)vertex_norms)[index]);
    

}

// rasterization with cpu-gpu load distribution. useful if system doesnt support nested kernels

triangle trianglesCPU[num_triangles];

void rasterizeTrianglesThread(const int threadId, const int trianglesToInit, const int start_idx) {
    for (int tind = start_idx; tind < start_idx + trianglesToInit; tind++) {
        triangle2D t2D = trianglesCPU[tind].convert_to_2D();
        triangle t = trianglesCPU[tind];
        const vec2 tmp = vec2(scr_w / 2, scr_h / 2);

        t2D.p1 = t2D.p1 + tmp;
        t2D.p2 = t2D.p2 + tmp;
        t2D.p3 = t2D.p3 + tmp;
        t2D.calc_denom_and_vals();

        triplevec3 tv3;
        hipMemcpyFromSymbol(&tv3, HIP_SYMBOL(vertex_norms), sizeof(triplevec3), tind * sizeof(triplevec3));

        fillPixels << <32, ((int)t2D.bound_box.max.x - (int)t2D.bound_box.min.x)* ((int)t2D.bound_box.max.y - (int)t2D.bound_box.min.y) / 32 + 1 >> > (t2D, vec3(t.p1.z, t.p2.z, t.p3.z), tv3);
    }
}

void rasterizeAllTrianglesCPUandGPU() {
    const int threadLimit = std::thread::hardware_concurrency() * 20;
    int triangles_called = 0;

    const int trianglesPerThread = num_triangles / threadLimit + 1;

    const int num_calls = num_triangles / threadLimit + 1;
    std::vector<std::thread> threads(num_calls);
    for (int c = 0; c < num_calls; c++) {
        int trianglesToRender = num_triangles < (triangles_called + trianglesPerThread) ? trianglesPerThread : num_triangles - triangles_called;
        threads[c] = std::thread(rasterizeTrianglesThread, c, trianglesToRender, triangles_called);
        triangles_called += trianglesToRender;
    }
    for (int c = 0; c < num_calls; c++) {
        threads[c].join();
    }
}

#define threads_rasterization 256

int clamp(int i) {
    return (i < max_streams) ? i : max_streams;
}

// being worked on
/*
void rasterize_all_triangles_multi_thread() {
    hipStream_t streams[max_streams];

    const int num_iterations = num_triangles / max_streams + 1;

    int total_tris = 0;

    for (int i = 0; i < num_iterations; i++) {
        const int num_streams = clamp((num_triangles - total_tris));
        total_tris += num_streams;
        for (int s = 0; s < num_streams; s++) {
            hipStreamCreate(&streams[s]);
            rasterize_triangle_multi_thread<<<512, >>>(s, color(1.0f, 0.0f, 0.0f));
        }
    }
}
*/

void rasterize_all_triangles(bool cpu_used) {
    float* d_depth_buffer;
    hipGetSymbolAddress((void**)&d_depth_buffer, depth_buffer);

    if (cpu_used) {
        rasterizeAllTrianglesCPUandGPU();
    }
    else {
        rasterize_triangles_single_thread << <threads_rasterization, num_triangles / threads_rasterization + 1 >> > ();
    }
    hipDeviceSynchronize();
    
    hipMemset(d_depth_buffer, 0, sizeof(float) * scr_w * scr_h);
}

void add_triangle(vec3 p1, vec3 p2, vec3 p3, int idx) {
    triangle t = triangle(p1, p2, p3);
    hipMemcpyToSymbol(HIP_SYMBOL(triangles), &t, sizeof(triangle), sizeof(triangle) * idx);
}

vec3 computeNorm(vec3 p1, vec3 p2, vec3 p3) {
    return cross(p1 - p2, p1 - p3).normalize();
}

void loadRawModel(const char* filename, const char* filenameNorm, int start_idx) {
    vec3* vertexNorms = (vec3*)calloc(num_triangles * 3, sizeof(vec3));

    vec3 verts[num_triangles * 3];
    vec3 facetNorms[num_triangles];

    std::ifstream file(filename, std::ios::binary);
    if (!file) {
        std::cerr << "Unable to open file: " << filename << std::endl;
        return;
    }

    int i = 0;
    while (file.good()) {
        vec3 vertices[3];

        file.read(reinterpret_cast<char*>(&vertices[0]), sizeof(vec3));
        file.read(reinterpret_cast<char*>(&vertices[1]), sizeof(vec3));
        file.read(reinterpret_cast<char*>(&vertices[2]), sizeof(vec3));

        for (int i2 = 0; i2 < 3; i2++) { float tmp = vertices[i2].z; vertices[i2].z = vertices[i2].y + 60.0f; vertices[i2].y = tmp; }

        verts[i * 3] = vertices[0];
        verts[i * 3 + 1] = vertices[1];
        verts[i * 3 + 2] = vertices[2];
        if (!file.eof()) {
            add_triangle(vertices[0], vertices[1], vertices[2], start_idx + i);
            i++;
        }
    }

    file.close();

    std::ifstream fileNorm(filenameNorm);

    if (!fileNorm) {
        std::cerr << "Unable to open normal vector file: " << filenameNorm << std::endl;
        return;
    }

    i = 0;
    std::string line;
    while (std::getline(fileNorm, line)) {
        std::stringstream ss(line);
        vec3 n;
        ss >> n.x >> n.y >> n.z;

        if (i < num_triangles) {
            facetNorms[i] = n.normalize();
            i++;
        }
        else {
            std::cerr << "More normals than expected. Stopping read." << std::endl;
            break;
        }
    }

    fileNorm.close();

    for (int n = 0; n < num_triangles*3; n++) {
        for (int v2 = 0; v2 < num_triangles * 3; v2++) {
            if (verts[v2] == verts[n]) {
                vertexNorms[v2] = vertexNorms[v2] + facetNorms[n/3];
            }
        }
    }

    for (int i = 0; i < num_triangles * 3; i++) {
        vertexNorms[i].normalize();
        //printf("%f %f %f\n", vertexNorms[i].x, vertexNorms[i].y, vertexNorms[i].z);
    }

    hipMemcpyToSymbol(HIP_SYMBOL(vertex_norms), vertexNorms, sizeof(vec3) * num_triangles * 3);
    free(vertexNorms);
}



//*****************************************************************************************************************************************************************************************
// opengl stuff
// draws 2 triangles at z=0 and textures them with the pixel colors outputted by the cuda program
// no interop, data transfers from GPU to CPU each frame

char cpu_colors[sizeof(color) * scr_w * scr_h];

void copyBufferToCPU() {
    hipMemcpyFromSymbol(cpu_colors, HIP_SYMBOL(screen_buffer), sizeof(color) * scr_w * scr_h);
}

void framebuffer_size_callback(GLFWwindow* window, int width, int height);
void processInput(GLFWwindow* window);

// settings
unsigned int SCR_WIDTH = scr_w;
unsigned int SCR_HEIGHT = scr_h;


char* vertexShaderSource = "#version 330 core\n"
"layout (location = 0) in vec3 aPos;\n"
"layout(location = 1) in vec3 aColor;\n"
"layout(location = 2) in vec2 aTexCoord;\n"
"out vec3 ourColor;\n"
"out vec2 TexCoord;\n"
"void main()\n"
"{\n"
"   gl_Position = vec4(aPos.x, aPos.y, aPos.z, 1.0);\n"
"   ourColor = aColor;\n"
"   TexCoord = vec2(aTexCoord.x, aTexCoord.y);\n"
"}\0";

char* fragmentShaderSource = "#version 330 core\n"
"out vec4 FragColor;\n"
"in vec3 ourColor;\n"
"in vec2 TexCoord;\n"
"uniform sampler2D texture1;\n"
"void main()\n"
"{\n"
"   FragColor = texture(texture1, TexCoord);\n"
"}\n\0";

float truncate(float f) {
    return fabs(1.0 / (1.0 + exp(-1.0 * f))-0.5) * 2.0f;
}

int main()
{
    // add triangles
    
    for (int i = 0; i < 1; i++) {
        loadRawModel("C:\\Users\\david\\Downloads\\pythonAndModels\\raw_model.raw", "C:\\Users\\david\\Downloads\\pythonAndModels\\norm_vecs.txt", i * 207);
    }

    hipMemcpyFromSymbol(trianglesCPU, HIP_SYMBOL(triangles), sizeof(triangle) * num_triangles);

    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
#ifdef __APPLE__
    glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
#endif
    GLFWmonitor* monitor = glfwGetPrimaryMonitor();
    const GLFWvidmode* mode = glfwGetVideoMode(monitor);

    GLFWwindow* window = glfwCreateWindow(scr_w, scr_h, "Cuda-openGL Interop", NULL, NULL);

    glfwMakeContextCurrent(window);
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
    gladLoadGLLoader((GLADloadproc)glfwGetProcAddress);
    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        return -1;
    }
    unsigned int vertexShader = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
    glCompileShader(vertexShader);
    // check for shader compile errors
    int success;
    char infoLog[512];
    glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
    unsigned int fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
    glCompileShader(fragmentShader);
    // check for shader compile errors
    glGetShaderiv(fragmentShader, GL_COMPILE_STATUS, &success);
    if (!success) {
        glGetProgramInfoLog(fragmentShader, 512, NULL, infoLog);
        printf("ERROR::FRAGMENT::PROGRAM::LINKING_FAILED %s\n", infoLog);
    }
    GLuint shaderProgram = glCreateProgram();
    glAttachShader(shaderProgram, vertexShader);
    glAttachShader(shaderProgram, fragmentShader);
    glLinkProgram(shaderProgram);
    // check for linking errors
    glGetProgramiv(shaderProgram, GL_LINK_STATUS, &success);
    if (!success) {
        glGetProgramInfoLog(shaderProgram, 512, NULL, infoLog);
        printf("ERROR::SHADER::PROGRAM::LINKING_FAILED %s\n", infoLog);
    }

    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    float vertices[] = {
        // positions          // colors           // texture coords
         1.0f,  1.0f, 0.0f,   1.0f, 1.0f, 1.0f,   1.0f, 1.0f, // top right
         1.0f, -1.0f, 0.0f,   1.0f, 1.0f, 1.0f,   1.0f, 0.0f, // bottom right
        -1.0f, -1.0f, 0.0f,   1.0f, 1.0f, 1.0f,   0.0f, 0.0f, // bottom left
        -1.0f,  1.0f, 0.0f,   1.0f, 1.0f, 1.0f,   0.0f, 1.0f  // top left 
    };
    unsigned int indices[] = {
        0, 1, 3, // first triangle
        1, 2, 3  // second triangle
    };
    unsigned int VBO, VAO, EBO;
    glGenVertexArrays(1, &VAO);
    glGenBuffers(1, &VBO);
    glGenBuffers(1, &EBO);

    glBindVertexArray(VAO);

    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);

	// position attribute
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);
	// color attribute
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void*)(3 * sizeof(float)));
	glEnableVertexAttribArray(1);
	// texture coord attribute
	glVertexAttribPointer(2, 2, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void*)(6 * sizeof(float)));
	glEnableVertexAttribArray(2);

    unsigned int texture1;
    //uint8_t pixels[grid_h * grid_l * 3];
    glPixelStorei(GL_UNPACK_ALIGNMENT, 1);
    glGenTextures(1, &texture1);
    glBindTexture(GL_TEXTURE_2D, texture1);
    // set the texture wrapping parameters
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);	
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST_MIPMAP_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST_MIPMAP_NEAREST);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, scr_w, scr_h, 0, GL_RGB, GL_FLOAT, cpu_colors);
	glGenerateMipmap(GL_TEXTURE_2D);
    glUseProgram(shaderProgram); 
    glUniform1i(glGetUniformLocation(shaderProgram, "texture1"), 0);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    int frametime = 0;
    unsigned int frame = 0;
    
    while (!glfwWindowShouldClose(window))
    {
        hipEventRecord(start);
        processInput(window);
        glClearColor(0.0f, 0.0f, 1.0f, 1.0f);
        //glClear(GL_COLOR_BUFFER_BIT);
        // **
        // dodaj boje tu u pixels
        for (int i = 0; i < 1; i++) {
            rasterize_all_triangles(false);
        }
        copyBufferToCPU();
        // **
		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, scr_w, scr_h, 0, GL_RGB, GL_FLOAT, cpu_colors);
		glGenerateMipmap(GL_TEXTURE_2D);
        // bind textures on corresponding texture units
        glActiveTexture(GL_TEXTURE0);
        glBindTexture(GL_TEXTURE_2D, texture1);

        // render container
        glUseProgram(shaderProgram);
        glBindVertexArray(VAO);
        glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);

        glfwSwapBuffers(window);
        glfwPollEvents();
        goto skp;
        if (frame % 10 == 1) {
            int fps = 10000 / (frametime);
            //printf("%d\n", fps);
            if (fps < 10) {
                printf("\rFPS: 000%d", fps);
            }
            else if (fps < 100) {
                printf("\rFPS: 00%d", fps);
            }
            else if (fps < 1000) {
                printf("\rFPS: 0%d", fps);
            }

            else {
                printf("\rFPS: %d", fps);
            }
            frametime = 0;
        }
        skp:
        
        hipEventRecord(end);
        frame++;
        float milis;
        hipEventElapsedTime(&milis, start, end);
        frametime += milis;
        printf("%f\n", milis);
        printLastErrorCUDA()
    }
    glDeleteVertexArrays(1, &VAO);
    glDeleteBuffers(1, &VBO);
    glDeleteBuffers(1, &EBO);
    glfwTerminate();
    return 0;
}

void processInput(GLFWwindow* window)
{
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);
}

void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}