#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <glad/glad.h>
#include <glfw3.h>
#include <string.h>

#define DEBUG //if defined, extra stuff is printed(shouldn't slow down program noticeably)

// init of structs and methods as well as global vars and respective functions and macros
//*****************************************************************************************************************************************************************************************

// sizes for cfd
#define grid_l 512
#define grid_h 512


// vector for 2d cfd
#pragma pack(push, 4) // seems optimal to me
struct vec2 {
    float x, y;

    __host__ __device__ vec2() : x(0.0f), y(0.0f) {}

    __host__ __device__ vec2(float X, float Y) : x(X), y(Y) {}

    inline __host__ __device__ vec2 operator+(const vec2& f) const {
        return vec2(x + f.x, y + f.y);
    }

    inline __host__ __device__ vec2 operator-(const vec2& f) const {
        return vec2(x - f.x, y - f.y);
    }

    inline __host__ __device__ vec2 operator*(const float scalar) const {
        return vec2(x * scalar, y * scalar);
    }
};
#pragma pop

// global pointers for storing vecs
//__device__ char vectors[(grid_l * (grid_h + 1) + grid_h * (grid_l + 1)) * sizeof(vec2)];
//__device__ char vectorBuffer[(grid_l * (grid_h + 1) + grid_h * (grid_l + 1)) * sizeof(vec2)]; // used for divergence change and advection storage
__device__ char* vectors;
__device__ char* vectorBuffer;

__device__ bool barrier[grid_l * grid_h];

// macros for accessing array
// * "top" of a cell is lower in index than "bottom"
// * "top" of a cell is positive(if positive, then contribution to divergence is positive)
// * "right" of cell is higher in index than "left"
// * "right" of a cell is positive(if positive, then contribution to divergence is positive)
// * "left" and "down" negatively contribute to divergence
#define numHorizontal ((grid_l+1) * grid_h)
#define numVertical ((grid_h+1)*grid_l)

#define horizontalVectors ((vec2*)vectors)
#define verticalVectors ((vec2*)(vectors + numHorizontal * sizeof(vec2)))

#define horizontalVectorsBuffer ((vec2*)vectorBuffer)
#define verticalVectorsBuffer ((vec2*)(vectorBuffer + numHorizontal * sizeof(vec2)))

#define horizontalVectorsCPU ((vec2*)cpuVecs)
#define verticalVectorsCPU ((vec2*)(cpuVecs+ numHorizontal * sizeof(vec2)))

#define rightVecIndex(cellX, cellY) (cellX + 1 + cellY * (grid_l + 1)) 
#define leftVecIndex(cellX, cellY) (cellX + cellY * (grid_l + 1))
#define upVecIndex(cellX, cellY) (cellX + cellY * grid_l)
#define downVecIndex(cellX, cellY) (cellX + (cellY+1) * grid_l)

#define verticalVecIndex(x, y) (x + y * (grid_l))
#define horizontalVecIndex(x, y) (x + y * (grid_l+1))

#define inVerticalBounds(x, y) (x >= 0 && x < grid_l && y >= 0 && y <= grid_h)
#define inHorizontalBounds(x, y) (x >= 0 && x <= grid_l && y >= 0 && y < grid_h)

#define inCellBounds(x, y) (x >= 0 && x < grid_l && y >= 0 && y < grid_h)

#define cellXFromPos(p) (int)p.x
#define cellYFromPos(p) (int)p.y

// init grid
inline __device__ void init_vec() {
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    const int x = id % grid_l;
    const int y = id / grid_l;

    // set all vals to 0
    horizontalVectors[rightVecIndex(x, y)] = vec2(1.0f, 1.0f);
    verticalVectors[upVecIndex(x, y)] = vec2(1.0f, 1.0f);
    //verticalVectors[downVecIndex(x, y)] = vec2();
    //horizontalVectors[leftVecIndex(x, y)] = vec2();
}

// sets both left and right vecs of cell to v
inline __device__ void set_horizontal_vec_cell(const vec2 v, const int x, const int y) {
    horizontalVectors[rightVecIndex(x, y)] = v;
    horizontalVectors[leftVecIndex(x, y)] = v;
}

// sets both up and down vecs of cell to v
inline __device__ void set_vertical_vec_cell(const vec2 v, const int x, const int y) {
    verticalVectors[upVecIndex(x, y)] = v;
    verticalVectors[downVecIndex(x, y)] = v;
}

// init barrier
inline __device__ void init_barrier() {
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    barrier[id] = false;
}

// set val for a single barrier
inline __device__ void set_barrier(const int x, const int y) {
    barrier[x + y * grid_l] = true;
}

__global__ void setBarrier(const int x, const int y) {
    set_barrier(x, y);
}

void setBar(const int x, const int y) {
    setBarrier << <1, 1 >> > (x, y);
}

// reset kernels
__global__ void resetVectors() {
    init_vec();
}

__global__ void resetBarriers() {
    init_barrier();
}

void resetVecs() {
    resetVectors << <512, grid_l * grid_h / 512 >> > ();
}

void resetBars() {
    resetBarriers << <512, grid_l * grid_h / 512 >> > ();
}

// copy kernels
__global__ void swapBuffer() {
    char* tmp = vectors;
    vectors = vectorBuffer;
    vectorBuffer = tmp;
}

// divergence functions and kernel
//*****************************************************************************************************************************************************************************************

// calcs divergence for a single cell
inline __device__ float calc_divergence(const int x, const int y) {
    return verticalVectors[upVecIndex(x, y)].y - verticalVectors[downVecIndex(x, y)].y + horizontalVectors[rightVecIndex(x, y)].x - horizontalVectors[leftVecIndex(x, y)].x;
}

#define L 0
#define R 1
#define T 2
#define B 3
inline __device__ void apply_divergence(const int x, const int y) {
    float divergence = calc_divergence(x, y);
    unsigned char num_affected = 0;
    bool affected_cells[4];
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        affected_cells[i] = false;
    }

    // get rights
    #pragma unroll
    for (int xo = -1; xo < 1; xo += 2) {
        const bool tmp = barrier[x + xo + y * grid_l];
        num_affected += tmp;
        affected_cells[(xo + 1) / 2] = tmp;
    }

    // get bottoms
    #pragma unroll
    for (int yo = -1; yo < 1; yo += 2) {
        const bool tmp = barrier[x + (y + yo) * grid_l];
        num_affected += tmp;
        affected_cells[(yo + 5) / 2] = tmp;
    }

    divergence /= num_affected;

    // subtract the divergence equally from each affected vector(not blocked by a barrier)
    verticalVectors[upVecIndex(x, y)].y -= divergence * affected_cells[T]; // up
    verticalVectors[downVecIndex(x, y)].y += divergence * affected_cells[B]; // down
    horizontalVectors[rightVecIndex(x, y)].y -= divergence * affected_cells[R]; // right
    horizontalVectors[leftVecIndex(x, y)].y -= divergence * affected_cells[L]; // left
}

// divergence equations are solved(variables eliminated) using gaussian elimination, and each iteration is done in 2 passes
// each pass is either "white" or "black", and these colors represent the squares on a checkerboard

// threads per block divergence
#define threads_divergence 512
#define blocks_divergence (grid_l * grid_h / 2) / threads_divergence + 1

// divergence kernel "white"
__global__ void divergenceGaussianW() {
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    const int cellId = id * 2; // account for checkerboard spacing
    apply_divergence(cellId % grid_l, cellId / grid_l); // may remove this func due to overhead
}

// divergence kernel "black"
__global__ void divergenceGaussianB() {
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    const int cellId = id * 2 + 1; // account for checkerboard spacing
    if (cellId >= grid_l * grid_h) { return; }
    apply_divergence(cellId % grid_l, cellId / grid_l); // may remove this func due to overhead
}

// cpu function to call kernels
void gaussianDivergenceSolver(const int passes) {
    for (int p = 0; p < passes; p++) {
        divergenceGaussianW << <threads_divergence, blocks_divergence >> > ();
        divergenceGaussianB << <threads_divergence, blocks_divergence >> > ();
    }
}

// advection functions and kernel
//*****************************************************************************************************************************************************************************************

inline __device__ vec2 sample_surrounding_vecs_H(const int x, const int y) {
    // gets the avg vertical component around a horizontal vector at (x, y). returns both the horizontal vector and the vertical component in a vec2
    const unsigned char num_verts = inVerticalBounds(x, y) + inVerticalBounds(x + 1, y) + inVerticalBounds(x, y + 1) + inVerticalBounds(x + 1, y + 1);// num of vertical vecs sampled

    const float vert_comp = (verticalVectors[verticalVecIndex(x, y)].y * inVerticalBounds(x, y) + verticalVectors[verticalVecIndex(x+1, y)].y * inVerticalBounds(x+1, y) + verticalVectors[verticalVecIndex(x, y+1)].y * inVerticalBounds(x, y+1) + verticalVectors[verticalVecIndex(x+1, y+1)].y * inVerticalBounds(x+1, y+1)) / num_verts;// bounds calcs redone to avoid branching(may be slower on some systems)
    
    return vec2(horizontalVectors[horizontalVecIndex(x, y)].x, vert_comp);
}

inline __device__ vec2 sample_surrounding_vecs_V(const int x, const int y) {
    // gets the avg horizontal component around a vertical vector at (x, y). returns both the horizontal vector and the vertical component in a vec2
    const unsigned char num_hors = inHorizontalBounds(x, y) + inHorizontalBounds(x + 1, y) + inHorizontalBounds(x, y - 1) + inHorizontalBounds(x + 1, y - 1);// num of vertical vecs sampled

    const float hor_comp = (horizontalVectors[horizontalVecIndex(x, y)].y * inHorizontalBounds(x, y) + horizontalVectors[horizontalVecIndex(x + 1, y)].y * inHorizontalBounds(x + 1, y) + horizontalVectors[horizontalVecIndex(x, y - 1)].y * inHorizontalBounds(x, y - 1) + horizontalVectors[horizontalVecIndex(x + 1, y - 1)].y * inHorizontalBounds(x + 1, y - 1)) / num_hors;// bounds calcs redone to avoid branching(may be slower on some systems)

    return vec2(hor_comp, verticalVectors[verticalVecIndex(x, y)].y);
}

inline __device__ vec2 get_previous_value_V(const int x, const int y) {
    // get the velocity value the vector comes from and advect it
    // semi lagrangian advection
    vec2 ret = verticalVectors[verticalVecIndex(x, y)];
    const vec2 previous_path = sample_surrounding_vecs_V(x, y) * -1;
    const vec2 pos = vec2(x + 0.5f, y) + previous_path;

    const int px = cellXFromPos(pos);
    const int py = cellYFromPos(pos);
    if (!inCellBounds(px, py)) { return ret; }

    ret.y = 0.0f; // gonna add to it later

    // right bottom cell
    const int cellxy_offset[] = {px+1, px-1, py+1, py-1};
    unsigned char num_sampled = 0;

    #pragma unroll
    for (int i = 0; i < 4; i++) {
        const unsigned char xo = (i < 2) * cellxy_offset[i]; // avoid branching and only have one loop
        const unsigned char yo = (i >= 2) * cellxy_offset[i];
        if (!inCellBounds(px + xo, py + yo)) { continue; } // branching unavoidable

        num_sampled += 2;
        ret.y += verticalVectors[upVecIndex(px + xo, py + yo)].y + verticalVectors[downVecIndex(px + xo, py + yo)].y;
    }
    ret.y /= num_sampled;
    return ret;
}

inline __device__ vec2 get_previous_value_H(const int x, const int y) {
    // get the velocity value the vector comes from and advect it
    // semi lagrangian advection
    vec2 ret = horizontalVectors[horizontalVecIndex(x, y)];
    const vec2 previous_path = sample_surrounding_vecs_H(x, y) * -1;
    const vec2 pos = vec2(x, y + 0.5f) + previous_path;

    const int px = cellXFromPos(pos);
    const int py = cellYFromPos(pos);
    if (!inCellBounds(px, py)) { return ret; }

    ret.x = 0.0f; // gonna add to it later

    // right bottom cell
    const int cellxy_offset[] = { px + 1, px - 1, py + 1, py - 1 };
    unsigned char num_sampled = 0;

#pragma unroll
    for (int i = 0; i < 4; i++) {
        const unsigned char xo = (i < 2) * cellxy_offset[i]; // avoid branching and only have one loop
        const unsigned char yo = (i >= 2) * cellxy_offset[i];
        if (!inCellBounds(px + xo, py + yo)) { continue; } // branching unavoidable

        num_sampled += 2;
        ret.x += horizontalVectors[rightVecIndex(px + xo, py + yo)].y + horizontalVectors[leftVecIndex(px + xo, py + yo)].y;
    }
    ret.x /= num_sampled;
    return ret;
}

__global__ void advectionKernel() {
    // advects 2-4 vectors instead of 1 vector to distribute workload amongst more threads
    const int id = threadIdx.x + blockIdx.x * blockDim.x;

    const int cx = id % grid_l;
    const int cy = id / grid_l;
    
    // buffer must be used to prevent reads and writes at the same time, and to keep results accurate
    horizontalVectorsBuffer[leftVecIndex(cx, cy)] = get_previous_value_H(cx, cy);
    verticalVectorsBuffer[upVecIndex(cx, cy)] = get_previous_value_V(cx, cy);

    // unavoidable branching(i think)
    if (cx == grid_l - 1) {
        horizontalVectorsBuffer[rightVecIndex(cx, cy)] = get_previous_value_H(cx+1, cy);
    }
    if (cy == grid_h - 1) {
        verticalVectorsBuffer[downVecIndex(cx, cy)] = get_previous_value_V(cx, cy+1);
    }
}

#define threads_advection 512
#define blocks_advection grid_l * grid_h / threads_advection
void semiLagrangianAdvection() {
    advectionKernel << <threads_advection, blocks_advection >> > ();
    swapBuffer << <1, 1 >> > ();
}

// alloc and mem moving functions
//*****************************************************************************************************************************************************************************************

char cpuVecs[(grid_l * (grid_h + 1) + grid_h * (grid_l + 1)) * sizeof(vec2)];
bool cpuBarrier[grid_l * grid_h];

char* deviceVecPointer;
char* deviceVecBufferPointer;

void allocDeviceVars() {
    // tmp cpu pointer used
    #ifdef DEBUG
    hipError_t m1, m2, c1, c2;
    m1 = hipMalloc((void**)(&deviceVecPointer), (grid_l * (grid_h + 1) + grid_h * (grid_l + 1)) * sizeof(vec2));
    c1 = hipMemcpyToSymbol(HIP_SYMBOL(vectors), &deviceVecPointer, sizeof(char*));
    
    m2 = hipMalloc((void**)(&deviceVecBufferPointer), (grid_l * (grid_h + 1) + grid_h * (grid_l + 1)) * sizeof(vec2));
    c2 = hipMemcpyToSymbol(HIP_SYMBOL(vectorBuffer), &deviceVecBufferPointer, sizeof(char*));

    printf("alloc one     malloc: %s | copy: %s\n", hipGetErrorString(m1), hipGetErrorString(c1));
    printf("alloc two     malloc: %s | copy: %s\n", hipGetErrorString(m2), hipGetErrorString(c2));
    #endif

    #ifndef DEBUG
    hipMalloc((void**)(&temp), (grid_l * (grid_h + 1) + grid_h * (grid_l + 1)) * sizeof(vec2));
    hipMemcpyToSymbol(HIP_SYMBOL(vectors), &temp, sizeof(char*));

    hipMalloc((void**)(&temp), (grid_l * (grid_h + 1) + grid_h * (grid_l + 1)) * sizeof(vec2));
    hipMemcpyToSymbol(HIP_SYMBOL(vectorBuffer), &temp, sizeof(char*));
    #endif
}

void moveMainArrayToCPU() {
    hipError_t e = hipMemcpy(cpuVecs, deviceVecPointer, (grid_l * (grid_h + 1) + grid_h * (grid_l + 1)) * sizeof(vec2), hipMemcpyDeviceToHost);
#ifdef DEBUG
    printf("copy vecs: %s\n", hipGetErrorString(e));
#endif
}

void moveBarrierToCPU() {
    hipError_t e = hipMemcpyFromSymbol(cpuBarrier, HIP_SYMBOL(barrier), grid_l * grid_h * sizeof(bool));
#ifdef DEBUG
    printf("copy barrier: %s\n", hipGetErrorString(e));
#endif
}

// sampling vector field for drawing
//*****************************************************************************************************************************************************************************************

struct color {
    unsigned char r, g, b;

    __host__ __device__ color() : r(0), g(0), b(0){}
    __host__ __device__ color(float red, float green, float blue) : r(red), g(green), b(blue) {}
};

color sampleFieldVelocityMagnitude(const int x, const int y, float threshold) {
    const float total = fabs(horizontalVectorsCPU[rightVecIndex(x, y)].x - horizontalVectorsCPU[leftVecIndex(x, y)].x + verticalVectorsCPU[upVecIndex(x, y)].y - verticalVectorsCPU[downVecIndex(x, y)].y);
    float magnitude = total / threshold;
    magnitude = (magnitude > 1.0f) ? 1.0f : magnitude;
    return color(magnitude * 255, 0, 0);
}

color sampleFieldVelocityDirectionalMagnitude(const int x, const int y, float threshold) {
    const float totalPos = fabs(horizontalVectorsCPU[rightVecIndex(x, y)].x + verticalVectorsCPU[upVecIndex(x, y)].y);
    const float totalNeg = fabs(horizontalVectorsCPU[leftVecIndex(x, y)].x + verticalVectorsCPU[downVecIndex(x, y)].y);

    float magnitudePos = totalPos / threshold;
    float magnitudeNeg = totalNeg / threshold;

    magnitudePos = (magnitudePos > 1.0f) ? 1.0f : magnitudePos;
    magnitudeNeg = (magnitudeNeg > 1.0f) ? 1.0f : magnitudeNeg;

    return color(magnitudePos * 255, magnitudeNeg * 255, 0);
}

unsigned char cpuColors[grid_l * grid_h];

void fillColorArray(float threshold, char* sampleType) {
    if (strcmp(sampleType, "magnitude") == 0) {
        for (int x = 0; x < grid_l; x++) {
            for (int y = 0; y < grid_h; y++) {
                ((color*)cpuColors)[x + y * grid_l] = sampleFieldVelocityMagnitude(x, y, threshold);
                if (cpuBarrier[x + y * grid_l]) {
                    ((color*)cpuColors)[x + y * grid_l] = color(255, 255, 255);
                }
            }
        }
    }

    if (strcmp(sampleType, "directional magnitude") == 0) {
        for (int x = 0; x < grid_l; x++) {
            for (int y = 0; y < grid_h; y++) {
                ((color*)cpuColors)[x + y * grid_l] = sampleFieldVelocityDirectionalMagnitude(x, y, threshold);
                if (cpuBarrier[x + y * grid_l]) {
                    ((color*)cpuColors)[x + y * grid_l] = color(255, 255, 255);
                }
            }
        }
    }
}

//*****************************************************************************************************************************************************************************************
// opengl stuff
// draws 2 triangles at z=0 and textures them with the pixel colors outputted by the cuda program
// no interop, data transfers from GPU to CPU and back to GPU each frame
void framebuffer_size_callback(GLFWwindow* window, int width, int height);
void processInput(GLFWwindow* window);

// settings
unsigned int SCR_WIDTH = grid_l;
unsigned int SCR_HEIGHT = grid_h;


char* vertexShaderSource = "#version 330 core\n"
"layout (location = 0) in vec3 aPos;\n"
"layout(location = 1) in vec3 aColor;\n"
"layout(location = 2) in vec2 aTexCoord;\n"
"out vec3 ourColor;\n"
"out vec2 TexCoord;\n"
"void main()\n"
"{\n"
"   gl_Position = vec4(aPos.x, aPos.y, aPos.z, 1.0);\n"
"   ourColor = aColor;\n"
"   TexCoord = vec2(aTexCoord.x, aTexCoord.y);\n"
"}\0";

char* fragmentShaderSource = "#version 330 core\n"
"out vec4 FragColor;\n"
"in vec3 ourColor;\n"
"in vec2 TexCoord;\n"
"uniform sampler2D texture1;\n"
"void main()\n"
"{\n"
"   FragColor = texture(texture1, TexCoord);\n"
"}\n\0";

float truncate(float f) {
    return fabs(1.0 / (1.0 + exp(-1.0 * f))-0.5) * 2.0f;
}

int main()
{
    // alloc and set all global device arrays/pointers to 0
    allocDeviceVars();
    resetVecs();
    resetBars();

    // for now, make a square barrier
    for (int x = 100; x < 200; x++) {
        for (int y = 200; y < 300; y++) {
            setBar(x, y);
        }
    }

    moveMainArrayToCPU();
    moveBarrierToCPU();

    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
#ifdef __APPLE__
    glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
#endif

    GLFWwindow* window = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "Cuda-openGL Interop", NULL, NULL);
    glfwMakeContextCurrent(window);
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
    gladLoadGLLoader((GLADloadproc)glfwGetProcAddress);
    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        return -1;
    }
    unsigned int vertexShader = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
    glCompileShader(vertexShader);
    // check for shader compile errors
    int success;
    char infoLog[512];
    glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
    unsigned int fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
    glCompileShader(fragmentShader);
    // check for shader compile errors
    glGetShaderiv(fragmentShader, GL_COMPILE_STATUS, &success);
    if (!success) {
        glGetProgramInfoLog(fragmentShader, 512, NULL, infoLog);
        printf("ERROR::FRAGMENT::PROGRAM::LINKING_FAILED %s\n", infoLog);
    }
    GLuint shaderProgram = glCreateProgram();
    glAttachShader(shaderProgram, vertexShader);
    glAttachShader(shaderProgram, fragmentShader);
    glLinkProgram(shaderProgram);
    // check for linking errors
    glGetProgramiv(shaderProgram, GL_LINK_STATUS, &success);
    if (!success) {
        glGetProgramInfoLog(shaderProgram, 512, NULL, infoLog);
        printf("ERROR::SHADER::PROGRAM::LINKING_FAILED %s\n", infoLog);
    }

    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    float vertices[] = {
        // positions          // colors           // texture coords
         1.0f,  1.0f, 0.0f,   1.0f, 1.0f, 1.0f,   1.0f, 1.0f, // top right
         1.0f, -1.0f, 0.0f,   1.0f, 1.0f, 1.0f,   1.0f, 0.0f, // bottom right
        -1.0f, -1.0f, 0.0f,   1.0f, 1.0f, 1.0f,   0.0f, 0.0f, // bottom left
        -1.0f,  1.0f, 0.0f,   1.0f, 1.0f, 1.0f,   0.0f, 1.0f  // top left 
    };
    unsigned int indices[] = {
        0, 1, 3, // first triangle
        1, 2, 3  // second triangle
    };
    unsigned int VBO, VAO, EBO;
    glGenVertexArrays(1, &VAO);
    glGenBuffers(1, &VBO);
    glGenBuffers(1, &EBO);

    glBindVertexArray(VAO);

    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);

	// position attribute
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);
	// color attribute
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void*)(3 * sizeof(float)));
	glEnableVertexAttribArray(1);
	// texture coord attribute
	glVertexAttribPointer(2, 2, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void*)(6 * sizeof(float)));
	glEnableVertexAttribArray(2);

    unsigned int texture1;
    //uint8_t pixels[grid_h * grid_l * 3];
    glPixelStorei(GL_UNPACK_ALIGNMENT, 1);
    glGenTextures(1, &texture1);
    glBindTexture(GL_TEXTURE_2D, texture1);
    // set the texture wrapping parameters
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);	
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST_MIPMAP_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, grid_l, grid_h, 0, GL_RGB, GL_UNSIGNED_BYTE, cpuColors);
	glGenerateMipmap(GL_TEXTURE_2D);
    glUseProgram(shaderProgram); 
    glUniform1i(glGetUniformLocation(shaderProgram, "texture1"), 0);
    while (!glfwWindowShouldClose(window))
    {
        processInput(window);
        glClearColor(0.0f, 0.0f, 1.0f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);
        int ind = 0;
        // **
        // dodaj boje tu u pixels
        fillColorArray(3.0f, "directional magnitude");
        // **
		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, grid_l, grid_h, 0, GL_RGB, GL_UNSIGNED_BYTE, cpuColors);
		glGenerateMipmap(GL_TEXTURE_2D);
        // bind textures on corresponding texture units
        glActiveTexture(GL_TEXTURE0);
        glBindTexture(GL_TEXTURE_2D, texture1);

        // render container
        glUseProgram(shaderProgram);
        glBindVertexArray(VAO);
        glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);

        glfwSwapBuffers(window);
        glfwPollEvents();
    }
    glDeleteVertexArrays(1, &VAO);
    glDeleteBuffers(1, &VBO);
    glDeleteBuffers(1, &EBO);
    glfwTerminate();
    return 0;
}

void processInput(GLFWwindow* window)
{
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);
}

void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}